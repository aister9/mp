#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"
#include <stdio.h>
#include <iostream>

#define LOOP_I(x) for(int i = 0 ; i<x; i++)

#define NUM_BLOCK 128*1024
#define NUM_T_IN_B 1024
#define ARRAY_SIZE NUM_T_IN_B*NUM_BLOCK

#define NUM_STREAMS 4

__global__ void myKernel(int *_in, int *_out) {
	int tID = blockDim.x * blockIdx.x + threadIdx.x;

	int temp = 0;
	for (int i = 0; i < 250; i++) {
		temp = (temp + _in[tID] * 5) % 10;
	}
	_out[tID] = temp;
}

void main() {
	DS_timer timer(5);
	timer.setTimerName(0, "Single Stream");
	timer.setTimerName(1, "*host -> device");
	timer.setTimerName(2, "*kernel execution");
	timer.setTimerName(3, "*devcie -> host");
	timer.setTimerName(4, "Multiple Stream");

	int *in = NULL, *out = NULL, *out2 = NULL, *dIn = NULL, *dOut = NULL;

	hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE); memset(in, 0, sizeof(int)*ARRAY_SIZE);
	hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE); memset(out, 0, sizeof(int)*ARRAY_SIZE);
	hipHostMalloc(&out2, sizeof(int)*ARRAY_SIZE); memset(out2, 0, sizeof(int)*ARRAY_SIZE);

	hipMalloc(&dIn, sizeof(int)*ARRAY_SIZE); hipMalloc(&dOut, sizeof(int)*ARRAY_SIZE);

	LOOP_I(ARRAY_SIZE) in[i] = rand() % 10;

	//single
	timer.onTimer(0);
	timer.onTimer(1);
	hipMemcpy(dIn, in, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
	timer.offTimer(1);
	timer.onTimer(2);
	myKernel << <NUM_BLOCK, NUM_T_IN_B >> > (dIn, dOut);
	hipDeviceSynchronize();
	timer.offTimer(2);
	timer.onTimer(3);
	hipMemcpy(out, dOut, sizeof(int)*ARRAY_SIZE, hipMemcpyDeviceToHost);
	timer.offTimer(3);
	timer.offTimer(0);
	hipStream_t stream[NUM_STREAMS];
	LOOP_I(NUM_STREAMS) hipStreamCreate(&stream[i]);

	int chunkSize = ARRAY_SIZE / NUM_STREAMS;

	timer.onTimer(4);
	LOOP_I(NUM_STREAMS) {
		int offset = chunkSize * i;
		hipEvent_t start, stop;
		hipEventCreate(&start); hipEventCreate(&stop);

		hipEventRecord(start);
		hipMemcpyAsync(dIn + offset, in + offset, sizeof(int)*chunkSize, hipMemcpyHostToDevice, stream[i]);
		myKernel << <NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i] >> > (dIn + offset, dOut + offset);
		hipMemcpyAsync(out2 + offset, dOut + offset, sizeof(int)*chunkSize, hipMemcpyDeviceToHost, stream[i]);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float time;
		hipEventElapsedTime(&time, start, stop);

		printf("Stream[%d] : %lf ms\n", i, time);

		hipEventDestroy(start); hipEventDestroy(stop);
	}
	hipDeviceSynchronize();
	timer.offTimer(4);

	LOOP_I(NUM_STREAMS) hipStreamDestroy(stream[i]);

	hipFree(dIn); hipFree(dOut);

	hipHostFree(in); hipHostFree(out); hipHostFree(out2);

	timer.printTimer();
	system("pause");
}