#include "hip/hip_runtime.h"
#include "DS_timer.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void threadCounting_noSync(int *a) {
	(*a)++;
}

__global__ void threadCounting_atomicGlobal(int *a) {
	atomicAdd(a, 1);
}

__global__ void threadCounting_atomicShared(int *a) {
	__shared__ int sa;

	if (threadIdx.x == 0) sa = 0;
	__syncthreads();

	atomicAdd(&sa, 1);
	__syncthreads();

	if (threadIdx.x == 0)
		atomicAdd(a, sa);
}


int main()
{
	DS_timer timer(3);
	timer.setTimerName(0, "NoAtomic");
	timer.setTimerName(1, "AtomicGlobal");
	timer.setTimerName(2, "AtomicShared");
	int a = 0; int *d1;
	int b = 0; int *d2;
	int c = 0; int *d3;

	hipMalloc((void **)&d1, sizeof(int));
	hipMalloc((void **)&d2, sizeof(int));
	hipMalloc((void **)&d3, sizeof(int));
	hipMemset(d1, 0, sizeof(int) * 1);
	hipMemset(d2, 0, sizeof(int) * 1);
	hipMemset(d3, 0, sizeof(int) * 1);

	timer.onTimer(0);
	threadCounting_noSync << <10240, 512 >> > (d1);
	hipDeviceSynchronize();
	timer.offTimer(0);


	timer.onTimer(1);
	threadCounting_atomicGlobal << <10240, 512 >> > (d2);
	hipDeviceSynchronize();
	timer.offTimer(1);

	timer.onTimer(2);
	threadCounting_atomicShared << <10240, 512 >> > (d3);
	hipDeviceSynchronize();
	timer.offTimer(2);

	hipMemcpy(&a, d1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&b, d2, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&c, d3, sizeof(int), hipMemcpyDeviceToHost);

	printf("[No Atomic ] : %d\n", a);
	printf("[Atomic Global] : %d\n", b);
	printf("[Atomic Shared] : %d\n", c);

	hipFree(d1);
	hipFree(d2);
	hipFree(d3);

	timer.printTimer();
}
